
#include <hip/hip_runtime.h>
extern "C" __global__ void julia(const unsigned int width,
                                 const unsigned int height,
                                 const float c_x,
                                 const float c_y,
                                 const int iterations,
                                 const float top,
                                 const float bottom,
                                 const float left,
                                 const float right,
                                 unsigned char *d_colors,
                                 const unsigned char *d_color_map) {
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idy = threadIdx.y + blockIdx.y * blockDim.y;

    // outside of image
    if (idx >= width && idy >= height) {
        return;
    }

    float x = left + (right - left) / (float) width * (float) idx;
    float y = top - (top - bottom) / (float) height * (float) idy;
    float x_temp;
    unsigned int steps;
    bool converged = false;

    // check if escaped
    for (int i = 0; i < iterations; i++) {
        if (x * x + y * y > 4) {
            steps = i % 256;
            converged = true;
            break;
        } else {
            x_temp = x * x - y * y;
            y = 2 * x * y + c_y;
            x = x_temp + c_x;
        }
    }

    // set colors
    unsigned int idc = (idy * width + idx) * 3;

    if (!converged) {
        d_colors[idc] = 0;
        d_colors[idc + 1] = 0;
        d_colors[idc + 2] = 0;
    } else {
        d_colors[idc] = d_color_map[steps * 3];
        d_colors[idc + 1] = d_color_map[steps * 3 + 1];
        d_colors[idc + 2] = d_color_map[steps * 3 + 2];
    }
}
