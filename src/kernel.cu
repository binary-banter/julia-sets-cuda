
#include <hip/hip_runtime.h>
extern "C" __global__ void julia(const int width,
                                 const int height,
                                 const float c_x,
                                 const float c_y,
                                 const int iterations,
                                 const float top,
                                 const float bottom,
                                 const float left,
                                 const float right,
                                 unsigned char *d_colors,
                                 const unsigned char *d_color_map) {
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if (idx < width && idy < height) {
        float x = left + (right - left) / (float) width * (float) idx;
        float y = top - (top - bottom) / (float) height * (float) idy;
        float x_temp;
        int steps = -1;

        // check if escaped
        for (int i = 0; i < iterations; i++) {
            if (x * x + y * y > 4) {
                steps = i;
                break;
            } else {
                x_temp = x * x - y * y;
                y = 2 * x * y + c_y;
                x = x_temp + c_x;
            }
        }

        // set colors
        unsigned int idc = (idy * width + idx) * 3;

        if (steps == -1) {
            d_colors[idc] = 0;
            d_colors[idc + 1] = 0;
            d_colors[idc + 2] = 0;
        } else {
            d_colors[idc] = d_color_map[steps * 3];
            d_colors[idc + 1] = d_color_map[steps * 3 + 1];
            d_colors[idc + 2] = d_color_map[steps * 3 + 2];
        }
    }
}